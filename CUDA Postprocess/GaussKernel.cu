#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include <stdlib.h>
#include <stdio.h>
#include "GaussKernel.h"

using namespace std;

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

texture <float, 2, hipReadModeElementType> tex1;

static hipArray *cuArray = NULL;

#define VERTICAL 0
#define HORIZONTAL 1

__global__ void gauss(float* output, int width, int height, int widthStep, float sigma, int direction, int matrixSize, int* matrix)
{	
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	matrix[0] = 1;

	int m3[] = { 1,2,1 };
	int m5[] = {1,4,6,4,1};
	if (matrixSize == 3)
		matrix = m3;
	else
		matrix = m5;
	int s = 0;

	for (int i = 0; i < matrixSize; i++)
	{
		s += matrix[i];
	}

	if (x >= widthStep || y >= widthStep) {
		return;
	}

	float outputValue = 0;

	if (direction == VERTICAL) {
		for (int j = 0; j < matrixSize; j++) {
			int x_offset = j - matrixSize / 2;
			outputValue += matrix[j] * tex2D(tex1, x + x_offset * sigma, y);
		}
		/*outputValue = 0.27901 * tex2D(tex1, x - sigma,y)
					+ 0.44198 * tex2D(tex1, x, y);
					+ 0.27901 *  tex2D(tex1, x + sigma, y);*/
	}
	else if (direction == HORIZONTAL) {
		for (int j = 0; j < matrixSize; j++) {
			int x_offset = j - matrixSize / 2;
			outputValue += matrix[j] * tex2D(tex1, x , y + x_offset * sigma);
		}
		}

	/*for (int i = 0; i < matrixSize; i++) {
		for (int j = 0; j < matrixSize; j++) {
			int x_offset = i, y_offset = j;
			x_offset -= matrixSize / 2;
			y_offset -= matrixSize / 2;
			outputValue += matrix[i*matrixSize + j] * tex2D(tex1, x + x_offset * sigma, y + y_offset * sigma);
		}
	}*/

	/*float outputValue = (matrix[0] * tex2D(tex1, x - sigma, y - sigma)) + (matrix[1] * tex2D(tex1, x, y - sigma)) + (matrix[2] * tex2D(tex1, x + sigma, y - sigma))
			+ (matrix[3] * tex2D(tex1, x - sigma, y)) + (matrix[4] * tex2D(tex1, x, y)) + (matrix[5] * tex2D(tex1, x + sigma, y))
			+ (matrix[6] * tex2D(tex1, x - sigma, y + sigma)) + (matrix[7] * tex2D(tex1, x, y + sigma)) + (matrix[8] * tex2D(tex1, x + sigma, y + sigma));
		*/
	//output[y*widthStep + x] = outputValue / s;
	output[y*widthStep + x] = outputValue/s;
}

inline void __cudaSafeCall(hipError_t err, const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
	if (hipSuccess != err)
	{
		printf("cudaSafeCall() failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}
#endif

	return;
}

void kernelGauss(float* input, float* output, int width, int height, int widthStep, float sigma, int direction, int matrixSize)
{
	int* matrix = (int*)malloc(sizeof(int)*matrixSize*matrixSize);
	matrix[0] = 1;

	/*for (int i = 1; i < matrixSize; i++)
	{
		if (i <= matrixSize / 2)
			matrix[i] = matrix[i - 1] * 2;
		else
			matrix[i] = matrix[i - 1] / 2;
	}

	for (int i = 1; i < matrixSize; i++) {
		for (int j = 0; j < matrixSize; j++) {
			if (i <= matrixSize / 2) {
				matrix[i*matrixSize + j] = matrix[((i - 1)*matrixSize) + j] * 2;
			}
			else {
				matrix[i*matrixSize + j] = matrix[((i - 1)*matrixSize) + j] / 2;
			}
		}
	}

	for (int i = 0; i < matrixSize; i++)
	{
		for (int j = 0; j < matrixSize; j++)
		{
			cout << matrix[i*matrixSize + j] << "\t";
		}
		cout << endl;
	}*/

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	CudaSafeCall(hipMallocArray(&cuArray, &channelDesc, width, height));

	hipMemcpy2DToArray(cuArray, 0, 0, input, widthStep, width * sizeof(float), height, hipMemcpyHostToDevice);

	hipBindTextureToArray(tex1, cuArray, channelDesc);

	float * D_output_x;
	int *D_matrix;
	CudaSafeCall(hipMalloc(&D_output_x, widthStep*height));
	CudaSafeCall(hipMalloc(&D_matrix, matrixSize*sizeof(int)));
	//hipMallocManaged(&D_matrix, sizeof(matrix), hipMemcpyHostToDevice);
	//memcpy(D_matrix, matrix, matrixSize*matrixSize*sizeof(int));
	dim3 blocksize(16, 16);
	dim3 gridsize;
	gridsize.x = (width + blocksize.x - 1) / blocksize.x;
	gridsize.y = (height + blocksize.y - 1) / blocksize.y;
	gauss << < gridsize, blocksize >> > (D_output_x, width, height, widthStep / sizeof(float), sigma, direction, matrixSize,D_matrix);
	hipDeviceSynchronize();

	hipUnbindTexture(tex1);

	CudaSafeCall(hipMemcpy(output, D_output_x, height*widthStep, hipMemcpyDeviceToHost));

	hipFree(D_output_x);
	hipFree(D_matrix);
	hipFree(matrix);
	hipFreeArray(cuArray);
	delete[] matrix;
}